#include "hip/hip_runtime.h"
// Aidan Draper
// CSC 431 - CUDA Kmeans Lab 3 Project
// November 28, 2018
// This is an application that parallelizes an original c file of a kmeans algorithm in CUDA.

#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <math.h>
#include <strings.h>
#include <time.h>
extern "C" { 
#include "gfx.h" 
}

typedef struct {
  int x;
  int y;
  int cluster;
} point_t;


typedef struct {
  point_t centroid;
  int size;
} set_t;

#define NO_CLUSTER -1
#define K 5

/* a way to choose colors */
unsigned int colors[] = { 0xFFFF00, 0x1CE6FF, 0xFF34FF, 0xFF4A46,
                          0x008941, 0x006FA6, 0xA30059, 0xFFDBE5, 0x7A4900,
                          0x0000A6, 0x63FFAC, 0xB79762, 0x004D43, 0x8FB0FF,
                          0x997D87, 0x5A0007, 0x809693, 0xFEFFE6, 0x1B4400,
                          0x4FC601, 0x3B5DFF, 0x4A3B53, 0xFF2F80, 0x61615A,
                          0xBA0900, 0x6B7900, 0x00C2A0, 0xFFAA92, 0xFF90C9,
                          0xB903AA, 0xD16100, 0xDDEFFF, 0x000035, 0x7B4F4B,
                          0xA1C299, 0x300018, 0x0AA6D8, 0x013349, 0x00846F,
                          0x372101, 0xFFB500, 0xC2FFED, 0xA079BF, 0xCC0744,
                          0xC0B9B2, 0xC2FF99, 0x001E09, 0x00489C, 0x6F0062,
                          0x0CBD66, 0xEEC3FF, 0x456D75, 0xB77B68, 0x7A87A1,
                          0x788D66, 0x885578, 0xFAD09F, 0xFF8A9A, 0xD157A0,
                          0xBEC459, 0x456648, 0x0086ED, 0x886F4C, 0x34362D,
                          0xB4A8BD, 0x00A6AA, 0x452C2C, 0x636375, 0xA3C8C9,
                          0xFF913F, 0x938A81, 0x575329, 0x00FECF, 0xB05B6F,
                          0x8CD0FF, 0x3B9700, 0x04F757, 0xC8A1A1, 0x1E6E00,
                          0x7900D7, 0xA77500, 0x6367A9, 0xA05837, 0x6B002C,
                          0x772600, 0xD790FF, 0x9B9700, 0x549E79, 0xFFF69F,
                          0x201625, 0x72418F, 0xBC23FF, 0x99ADC0, 0x3A2465,
                          0x922329, 0x5B4534, 0xFDE8DC, 0x404E55, 0x0089A3,
                          0xCB7E98, 0xA4E804, 0x324E72, 0x6A3A4C };

__global__ void assign(int num_observations, int size, point_t *dev_observations, set_t *dev_cluster){
     
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	if (tid < num_observations) {
		double min_mean = size * size;
		int min_cluster = NO_CLUSTER;
		for (int k = 0; k < K; k++) {
			double mean = powf(dev_observations[tid].x - dev_cluster[k].centroid.x, 2)
					+ powf(dev_observations[tid].y - dev_cluster[k].centroid.y, 2);
			if (min_mean > mean) {
				min_mean = mean;
				min_cluster = k;
			}
		}
		dev_observations[tid].cluster = min_cluster;
	}
}

/* draw the observations */
void show_observations(int num_observations, point_t *observation, set_t *cluster) {
   gfx_clear();

   // show the observations
   int j, i;
   for (j = 0; j < K; j++) {
      // change to the set color
      for (i = 0; i < num_observations; i++) {
         if (observation[i].cluster == -1) {
            gfx_color(255, 255, 255);
         } else {
            gfx_color((colors[observation[i].cluster] >> 16) & 0xFF,
                      (colors[observation[i].cluster] >> 8) & 0xFF,
                      colors[observation[i].cluster] & 0xFF);
         }
         gfx_line(observation[i].x, observation[i].y, observation[i].x, observation[i].y);
      }
   }

   for (i = 0; i < K; i++) {
      gfx_color((colors[i] >> 16) & 0xFF,
                (colors[i] >> 8) & 0xFF,
                colors[i] & 0xFF);

      gfx_line(cluster[i].centroid.x, cluster[i].centroid.y - 10, cluster[i].centroid.x, cluster[i].centroid.y + 10);
      gfx_line(cluster[i].centroid.x - 10, cluster[i].centroid.y, cluster[i].centroid.x + 10, cluster[i].centroid.y);
   }

   gfx_flush();
}

int main() {
	
  // only need to malloc structs and arrays

  /* read first line to determine how much data */
  int size, num_observations;
  scanf("%d %d", &num_observations, &size);

  /* data */
  point_t observations[num_observations];
  set_t cluster[K];

  /* read the data in */
  for (int i = 0; i < num_observations; i++) {
    scanf("%d %d", &(observations[i].x), &(observations[i].y));
    observations[i].cluster = NO_CLUSTER;
  }

  /* randomly set centroids */
  unsigned int seed = (unsigned int) time(NULL);
  for (int i = 0; i < K; i++) {
    int which = rand_r(&seed) % num_observations;

    // Forgy's Method
    cluster[i].centroid.x = observations[which].x;
    cluster[i].centroid.y = observations[which].y;

    cluster[i].size = 0;
    cluster[i].centroid.cluster = i;
  }
	
	point_t *dev_observations;
	set_t *dev_cluster;	

  // allocate memory and fill cluster and observation variables for CUDA
	hipMalloc(&dev_observations, num_observations * sizeof(point_t));
	hipMalloc(&dev_cluster,  K * sizeof(set_t));
	hipMemcpy(dev_observations, &observations, num_observations * sizeof(point_t), hipMemcpyHostToDevice);
	hipMemcpy(dev_cluster, &cluster, K * sizeof(set_t), hipMemcpyHostToDevice);

  gfx_open(size, size, "k-means clustering");
   
  set_t last_cluster[K]; // keeps track of our last cluster and centroids
  int centroid_move = 1; // a boolean value for checking convergence
  int iterations = 0;

  while (centroid_move > 0) {

    show_observations(num_observations, observations, cluster);

    // set all cluster sizes to 0
    for (int k = 0; k < K; k++) {
       cluster[k].size = 0;
    }

    iterations++;
    printf("Iteration %d\n", iterations);

    // Assignment step in cuda
    int thread_size = 256;
    int block_size = (num_observations + thread_size - 1)/ thread_size;
    
    assign<<<block_size,thread_size>>>(num_observations, size, dev_observations, dev_cluster);

    // Copy back down the assignment step
    hipMemcpy(observations, dev_observations, num_observations * sizeof(point_t), hipMemcpyDeviceToHost);
    hipMemcpy(cluster, dev_cluster, K * sizeof(set_t), hipMemcpyDeviceToHost);	

    // Update Step (calculate new centroids)
    if(iterations > 1){ // if its the second run or later, start recording the previous cluster
      memcpy(last_cluster, cluster, K * sizeof(set_t));
    }
    int sum_x[K], sum_y[K];
    bzero(sum_x, K * sizeof(int));
    bzero(sum_y, K * sizeof(int));

    //reset cluster size before checking it in the next loop
    for(int k = 0; k < K; k++){
      cluster[k].size = 0;
    }

    for (int i = 0; i < num_observations; i++) {
      cluster[observations[i].cluster].size++; // update size of each cluster as you go
      sum_x[observations[i].cluster] += observations[i].x; 
      sum_y[observations[i].cluster] += observations[i].y;
    }

    for (int k = 0; k < K; k++) { 
  printf("%d: sum_x = %d, sum_y = %d, cluster.size = %d\n", k, sum_x[k], sum_y[k], cluster[k].size);
       cluster[k].centroid.x = sum_x[k] / cluster[k].size;
       cluster[k].centroid.y = sum_y[k] / cluster[k].size;
       printf("cluster %d: (%d, %d)\n", k, cluster[k].centroid.x, cluster[k].centroid.y);
    }

  // Update CUDA's Versions of observations and cluster
  hipMemcpy(dev_observations, &observations, num_observations * sizeof(point_t), hipMemcpyHostToDevice);
  hipMemcpy(dev_cluster, &cluster, K * sizeof(set_t), hipMemcpyHostToDevice);

  // after first iteration, check for convergence
  if (iterations > 1){
    centroid_move = 0; // if all are same this will not change
    for (int k = 0; k < K; k++) {
      // if centroids have not changed, WE CONVERGED!
      if (last_cluster[k].centroid.x != cluster[k].centroid.x || last_cluster[k].centroid.y != cluster[k].centroid.y){
        centroid_move = 1;
      }
    }
  }

    sleep(1);
  }

  printf("Done with %d itertions\n", iterations);

  while (1) {
    char c = gfx_wait();
    if (c == 'q') break;
  }
}

// ssh -Y draper@10.52.1.13
/* gcc -o kmeans kmeans.c gfx.c -I/usr/X11/include -L/usr/X11/lib -lX11 -lm */
